#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>

#define WITH_UVM

#ifdef WITH_OPENGL
#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif
#endif

#include <hip/hip_runtime_api.h>

#ifdef WITH_OPENGL
#include <cuda_gl_interop.h>
#endif

//1 Coalescing
//2 Occupancy
//3 Shared Memory
//4 Texture Cache
//5 B


// STEP 0x00: Default code
// STEP 0x1a: Change the block size to 32x2 (improve coalescing) -- It. 1, Eclipse Ed.
// STEP 0x1b: Change the block size to 8x16 (improve occupancy) -- It. 1, Visual Studio Ed.
// STEP 0x20: Change the block size to 32x4 (improve occupancy)
// STEP 0x30: Use launch_bounds to register pressure (improve occupancy) -- It. 2, Visual Studio Ed.
// STEP 0x40: Use shared memory (improve memory accesses)
// STEP 0x50: Use read-only path (reduce pressure on Load-store unit)
// STEP 0x5a: Optimized convolution filter 2D
// STEP 0x60: Implement a separable filter (reduce arithmetic intensity)
// STEP 0x70: Process two elements per thread (improve memory efficiency, increase ILP)
// STEP 0x80: Improve shared memory accesses (reduce bank conflicts)
// STEP 0x90: Use floats rather than ints (reduce pressure on arithmetic pipe)
// STEP 0x91: Use floats and math intrinsics in sobel_filter (compile with --use_fast_math)
// STEP

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define CHECK(cond) do { \
  if( !(cond) ) { \
    fprintf(stderr, "Error at line %d in %s\n", __LINE__, __FILE__); \
    exit(1); \
  } \
} while(0)

#define CHECK_WITH_MSG(cond, msg) do { \
  if( !(cond) ) { \
    fprintf(stderr, "Error at line %d in %s: %s\n", __LINE__, __FILE__, msg); \
    exit(1); \
  } \
} while(0)

#define CHECK_CUDA(call) do { \
  hipError_t status = call; \
  if( status != hipSuccess ) { \
    fprintf(stderr, "CUDA Error at line %d in %s: %s\n", __LINE__, __FILE__, hipGetErrorString(status)); \
    exit(1); \
  } \
} while(0)

#define CHECK_OPENGL(call) do { \
  call; \
  GLenum status = glGetError(); \
  if( status != GL_NO_ERROR ) { \
    fprintf(stderr, "OpenGL Error at line %d in %s: %d\n", __LINE__, __FILE__, (int) status); \
    exit(1); \
  } \
} while(0)

double getElapsedTimeInMS(hipEvent_t eStart, hipEvent_t eStop) {
  CHECK_CUDA(hipEventSynchronize(eStop));
  float millisec = 0;
  hipEventElapsedTime(&millisec, eStart, eStop);
  return (double)millisec;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

typedef unsigned char uchar;
typedef unsigned int  uint;

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

enum { SHOW_RGBA = 0, SHOW_GRAYSCALE = 1, SHOW_SMOOTHED_GRAYSCALE = 2, SHOW_EDGES = 4 };

struct GlobalData
{
  int img_w;
  int img_h;

  uchar4 *img_rgba;
  uchar  *img_grayscale;
  uchar  *img_smoothed_grayscale;

#ifdef WITH_OPENGL
  hipGraphicsResource *img_cuda_pbo; 

  GLuint img_pbo; 
  GLuint img_tex;
#endif

  // What does the OpenGL renderer shows? 
  int show;
};

static GlobalData g_data;

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

static inline int round_up(int x, int y)
{
  return (x + y-1) / y;
}

// ====================================================================================================================

static __device__ __forceinline__ int in_img(int x, int y, int w, int h)
{
  return x >= 0 && x < w && y >= 0 && y < h;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

static __constant__ int gaussian_filter[7][7] = {
  1,   2,   3,    4,    3,    2,    1,
  2,   4,   6,    8,    6,    4,    2,
  3,   6,   9,   12,    9,    6,    3,
  4,   8,  12,   16,   12,    8,    4,
  3,   6,   9,   12,    9,    6,    3,
  2,   4,   6,    8,    6,    4,    2,
  1,   2,   3,    4,    3,    2,    1
};

static __constant__ float gaussian_filter_fp32[7] = {
  1.0f, 2.0f, 3.0f, 4.0f, 3.0f, 2.0f, 1.0f
};

static __constant__ int sobel_filter_x[3][3] = {
  -1, 0, 1,
  -2, 0, 2,
  -1, 0, 1,
};

static __constant__ int sobel_filter_y[3][3] = {
   1,  2,  1,
   0,  0,  0,
  -1, -2, -1,
};

static __constant__ float sobel_filter_v1_x[3][3] = {
  -1.0f, 0.0f, 1.0f,
  -2.0f, 0.0f, 2.0f,
  -1.0f, 0.0f, 1.0f,
};

static __constant__ float sobel_filter_v1_y[3][3] = {
   1.0f,  2.0f,  1.0f,
   0.0f,  0.0f,  0.0f,
  -1.0f, -2.0f, -1.0f,
};

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void rgba_to_grayscale_kernel_v0(int w, int h, const uchar4 *src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Early exit if the thread is not in the image.
  if( !in_img(x, y, w, h) )
    return;

  // Load the pixel in RGBA format.
  uchar4 p = src[y*w + x];

  // Extract the 3 components in FP32.
  float r = (float) p.x;
  float g = (float) p.y;
  float b = (float) p.z;

  // Compute the grayscale value.
  float gray = 0.298839f*r + 0.586811f*g + 0.114350f*b;

  // Store the result.
  dst[y*w + x] = (uchar) (gray >= 255.f ? 255 : gray);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void rgba_to_grayscale_kernel_v1(int w, int h, const uchar4 *src, uchar *dst)
{
  //compute two pixels at the same time
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = 2 * (blockIdx.y*blockDim.y) + threadIdx.y;

  // Load the pixel in RGBA format.
  uchar4 p0 = in_img(x, y, w, h) ? src[y*w + x] : make_uchar4(0,0,0,0);
  uchar4 p1 = in_img(x, y, w, h) ? src[(y+1)*w + x] : make_uchar4(0,0,0,0);

  // Extract the 3 components in FP32.
  float r0 = (float) p0.x;
  float g0 = (float) p0.y;
  float b0 = (float) p0.z;

  float r1 = (float) p1.x;
  float g1 = (float) p1.y;
  float b1 = (float) p1.z;

  // Compute the grayscale value.
  float gray0 = 0.298839f*r0 + 0.586811f*g0 + 0.114350f*b0;
  float gray1 = 0.298839f*r1 + 0.586811f*g1 + 0.114350f*b1;

  // Store the result.
  if(in_img(x, y, w, h))
    dst[y*w + x] = (uchar) (gray0 >= 255.f ? 255 : gray0);
  if(in_img(x, y+1, w, h))
    dst[(y+1)*w + x] = (uchar) (gray1 >= 255.f ? 255 : gray1);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gaussian_filter_7x7_v0(int w, int h, const uchar *src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Early exit if the thread is not in the image.
  if( !in_img(x, y, w, h) )
    return;

  // Load the 48 neighbours and myself.
  int n[7][7];
  for( int j = -3 ; j <= 3 ; ++j )
    for( int i = -3 ; i <= 3 ; ++i )
      n[j+3][i+3] = in_img(x+i, y+j, w, h) ? (int) src[(y+j)*w + (x+i)] : 0;

  // Compute the convolution.
  int p = 0;
  for( int j = 0 ; j < 7 ; ++j )
    for( int i = 0 ; i < 7 ; ++i )
      p += gaussian_filter[j][i] * n[j][i];

  // Store the result.
  dst[y*w + x] = (uchar) (p / 256);
}

// ====================================================================================================================

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 300
__global__ __launch_bounds__(128, 10) 
#elif defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 200
__global__ __launch_bounds__(128, 8) 
#else
__global__
#endif
void gaussian_filter_7x7_v1(int w, int h, const uchar *src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Early exit if the thread is not in the image.
  if( !in_img(x, y, w, h) )
    return;

  // Load the 48 neighbours and myself.
  int n[7][7];
  for( int j = -3 ; j <= 3 ; ++j )
    for( int i = -3 ; i <= 3 ; ++i )
      n[j+3][i+3] = in_img(x+i, y+j, w, h) ? (int) src[(y+j)*w + (x+i)] : 0;

  // Compute the convolution.
  int p = 0;
  for( int j = 0 ; j < 7 ; ++j )
    for( int i = 0 ; i < 7 ; ++i )
      p += gaussian_filter[j][i] * n[j][i];

  // Store the result.
  dst[y*w + x] = (uchar) (p / 256);
}

// ====================================================================================================================

__global__ void gaussian_filter_7x7_v2(int w, int h, const uchar *src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Shared memory.
  __shared__ uchar smem_img[10][64];

  // Load pixels to SMEM.
  uchar *smem_img_ptr = &smem_img[threadIdx.y][threadIdx.x];
  for( int iy = y-3 ; iy <= blockIdx.y*blockDim.y+6 ; iy += 4, smem_img_ptr += 4*64 )
  {
    smem_img_ptr[ 0] = in_img(x- 3, iy, w, h) ? src[iy*w + (x -3)] : 0;
    smem_img_ptr[32] = in_img(x+29, iy, w, h) ? src[iy*w + (x+29)] : 0; // 29 = 32-3.
  }
  __syncthreads();

  // Load the 48 neighbours and myself.
  int n[7][7];
  for( int j = 0 ; j <= 6 ; ++j )
    for( int i = 0 ; i <= 6 ; ++i )
      n[j][i] = smem_img[threadIdx.y+j][threadIdx.x+i];

  // Compute the convolution.
  int p = 0;
  for( int j = 0 ; j < 7 ; ++j )
    for( int i = 0 ; i < 7 ; ++i )
      p += gaussian_filter[j][i] * n[j][i];

  // Store the result.
  if( in_img(x, y, w, h) )
    dst[y*w + x] = (uchar) (p / 256);
}

// ====================================================================================================================

__global__ void gaussian_filter_7x7_v3(int w, int h, const uchar *__restrict src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Shared memory.
  __shared__ uchar smem_img[10][64];

  // Load pixels to SMEM.
  uchar *smem_img_ptr = &smem_img[threadIdx.y][threadIdx.x];
  for( int iy = y-3 ; iy <= blockIdx.y*blockDim.y+6 ; iy += 4, smem_img_ptr += 4*64 )
  {
    smem_img_ptr[ 0] = in_img(x- 3, iy, w, h) ? src[iy*w + (x -3)] : 0;
    smem_img_ptr[32] = in_img(x+29, iy, w, h) ? src[iy*w + (x+29)] : 0; // 29 = 32-3.
  }
  __syncthreads();

  // Load the 49 neighbours and myself.
  int n[7][7];
  for( int j = 0 ; j <= 6 ; ++j )
    for( int i = 0 ; i <= 6 ; ++i )
      n[j][i] = smem_img[threadIdx.y+j][threadIdx.x+i];

  // Compute the convolution.
  int p = 0;
  for( int j = 0 ; j < 7 ; ++j )
    for( int i = 0 ; i < 7 ; ++i )
      p += gaussian_filter[j][i] * n[j][i];

  // Store the result.
  if( in_img(x, y, w, h) )
    dst[y*w + x] = (uchar) (p / 256);
}

// ====================================================================================================================

__global__ void gaussian_filter_7x7_v3_bis(int w, int h, const uchar *__restrict src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = 1*(blockIdx.x*blockDim.x) + threadIdx.x;
  const int y = 2*(blockIdx.y*blockDim.y) + threadIdx.y;

  // Shared memory.
  __shared__ float smem_img[32][40];

  // Pixel to load.
  const int load_x = blockIdx.x*blockDim.x + 2*threadIdx.x - 4; // -4 for alignment (it should be -3).
  
  // Each thread loads 8 pixels.
  uchar2 p0 = in_img(load_x, y- 3, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y- 3)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p1 = in_img(load_x, y+ 5, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+ 5)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p2 = in_img(load_x, y+13, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+13)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p3 = in_img(load_x, y+21, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+21)*w + load_x]) : make_uchar2(0, 0);

  // Store to shared memory.
  if( threadIdx.x < 20 )
  {
    reinterpret_cast<float2*>(smem_img[threadIdx.y+ 0])[threadIdx.x] = make_float2((float) p0.x, (float) p0.y);
    reinterpret_cast<float2*>(smem_img[threadIdx.y+ 8])[threadIdx.x] = make_float2((float) p1.x, (float) p1.y);
    reinterpret_cast<float2*>(smem_img[threadIdx.y+16])[threadIdx.x] = make_float2((float) p2.x, (float) p2.y);
    reinterpret_cast<float2*>(smem_img[threadIdx.y+24])[threadIdx.x] = make_float2((float) p3.x, (float) p3.y);
  }
  __syncthreads();

  // Load the 49 neighbours and myself.
  float n[8][7];
  for( int j = 0 ; j <= 7 ; ++j )
    for( int i = 0 ; i <= 6 ; ++i )
      n[j][i] = smem_img[2*threadIdx.y+j][threadIdx.x+i];

  // Compute the convolutions.
  float p[2] = {0.0f};
  for( int j = 0 ; j < 7 ; ++j )
    for( int i = 0 ; i < 7 ; ++i )
    {
      p[0] += gaussian_filter[j][i] * n[j+0][i];
      p[1] += gaussian_filter[j][i] * n[j+1][i];
    }

  // Where to write the result 2*(blockIdx.x*blockDim.x + threadIdx.y).
  const int write_y = y + threadIdx.y;

  // Write the pixels.
  if( in_img(x, write_y, w, h) )
    dst[write_y*w + x] = (uchar) ((int) p[0] >> 8);
  if( in_img(x, write_y+1, w, h) )
    dst[(write_y+1)*w + x] = (uchar) ((int) p[1] >> 8);
}

// ====================================================================================================================

__global__ void gaussian_filter_7x7_v4(int w, int h, const uchar *__restrict src, uchar *dst) // 32x8 blocks.
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Shared memory.
  __shared__ int smem_img[16][38];

  // Each thread loads 2 pixels.
  int p0 = in_img(x-3, y-3, w, h) ? src[(y-3)*w + x-3] : 0;
  int p1 = in_img(x-3, y+5, w, h) ? src[(y+5)*w + x-3] : 0;
  
  // Load extra pixels per row.
  int p2 = in_img(x+29, y-3, w, h) && threadIdx.x < 6 ? src[(y-3)*w + x+29] : 0;
  int p3 = in_img(x+29, y+5, w, h) && threadIdx.x < 6 ? src[(y+5)*w + x+29] : 0;

  // Store to shared memory.
  smem_img[threadIdx.y+0][threadIdx.x] = p0;
  smem_img[threadIdx.y+8][threadIdx.x] = p1;

  // Store extra pixels.
  if( threadIdx.x < 6 )
  {
    smem_img[threadIdx.y+0][threadIdx.x+32] = p2;
    smem_img[threadIdx.y+8][threadIdx.x+32] = p3;
  }
  __syncthreads();
  
  // Compute the horizontal convolution.
  int n0[7], n1[7];
  for( int i = 0 ; i < 7 ; ++i )
  {
    n0[i] = smem_img[threadIdx.y+0][threadIdx.x+i];
    n1[i] = smem_img[threadIdx.y+8][threadIdx.x+i];
  }
  int p[2] = {0};
  for( int i = 0 ; i < 7 ; ++i )
  {
    p[0] += gaussian_filter[0][i] * n0[i];
    p[1] += gaussian_filter[0][i] * n1[i];
  }
  __syncthreads();

  // Write the result back to shared memory.
  smem_img[threadIdx.y+0][threadIdx.x] = p[0];
  smem_img[threadIdx.y+8][threadIdx.x] = p[1];

  // Make sure the results are in SMEM.
  __syncthreads();

  // Compute the vertical convolution.
  int n[7];
  for( int i = 0 ; i < 7 ; ++i )
    n[i] = smem_img[threadIdx.y+i][threadIdx.x];
  int q = 0;
  for( int i = 0 ; i < 7 ; ++i )
    q += gaussian_filter[i][0] * n[i];

  // Write the pixels.
  if( in_img(x, y, w, h) )
    dst[y*w + x] = (uchar) (q >> 8);
}

// ====================================================================================================================

__global__ void gaussian_filter_7x7_v5(int w, int h, const uchar *__restrict src, uchar *dst) // 32x8 blocks.
{
  // Position of the thread in the image.
  const int x = 1*(blockIdx.x*blockDim.x) + threadIdx.x;
  const int y = 2*(blockIdx.y*blockDim.y) + threadIdx.y;

  // Shared memory.
  __shared__ int smem_img[32][40];

  // Pixel to load.
  const int load_x = blockIdx.x*blockDim.x + 2*threadIdx.x - 4; // -4 for alignment (it should be -3).
  
  // Each thread loads 8 pixels.
  uchar2 p0 = in_img(load_x, y- 3, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y- 3)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p1 = in_img(load_x, y+ 5, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+ 5)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p2 = in_img(load_x, y+13, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+13)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p3 = in_img(load_x, y+21, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+21)*w + load_x]) : make_uchar2(0, 0);

  // Store to shared memory.
  if( threadIdx.x < 20 )
  {
    reinterpret_cast<int2*>(smem_img[threadIdx.y+ 0])[threadIdx.x] = make_int2(p0.x, p0.y);
    reinterpret_cast<int2*>(smem_img[threadIdx.y+ 8])[threadIdx.x] = make_int2(p1.x, p1.y);
    reinterpret_cast<int2*>(smem_img[threadIdx.y+16])[threadIdx.x] = make_int2(p2.x, p2.y);
    reinterpret_cast<int2*>(smem_img[threadIdx.y+24])[threadIdx.x] = make_int2(p3.x, p3.y);
  }
  __syncthreads();
  
  // Compute the horizontal convolution.
  int n0[7], n1[7], n2[7], n3[7];
  for( int i = 0 ; i < 7 ; ++i )
  {
    n0[i] = smem_img[threadIdx.y+ 0][threadIdx.x + i+1]; // +1 because of alignment constraint when loading pixels.
    n1[i] = smem_img[threadIdx.y+ 8][threadIdx.x + i+1];
    n2[i] = smem_img[threadIdx.y+16][threadIdx.x + i+1];
    n3[i] = smem_img[threadIdx.y+24][threadIdx.x + i+1];
  }
  int p[4] = {0};
  for( int i = 0 ; i < 7 ; ++i )
  {
    p[0] += gaussian_filter[0][i] * n0[i];
    p[1] += gaussian_filter[0][i] * n1[i];
    p[2] += gaussian_filter[0][i] * n2[i];
    p[3] += gaussian_filter[0][i] * n3[i];
  }
  __syncthreads();

  // Write the result back to shared memory.
  smem_img[threadIdx.y+ 0][threadIdx.x] = p[0];
  smem_img[threadIdx.y+ 8][threadIdx.x] = p[1];
  smem_img[threadIdx.y+16][threadIdx.x] = p[2];
  smem_img[threadIdx.y+24][threadIdx.x] = p[3];

  // Make sure the results are in SMEM.
  __syncthreads();

  // Compute the vertical convolution.
  int n[8];
  for( int i = 0 ; i < 8 ; ++i )
    n[i] = smem_img[2*threadIdx.y+i][threadIdx.x];
  int q0 = 0, q1 = 0;
  for( int i = 0 ; i < 7 ; ++i )
  {
    q0 += gaussian_filter[i][0] * n[i+0];
    q1 += gaussian_filter[i][0] * n[i+1];
  }

  // Where to write the result 2*(blockIdx.x*blockDim.x + threadIdx.y).
  const int write_y = y + threadIdx.y;

  // Write the pixels.
  if( in_img(x, write_y, w, h) )
    dst[write_y*w + x] = (uchar) (q0 >> 8);
  if( in_img(x, write_y+1, w, h) )
    dst[(write_y+1)*w + x] = (uchar) (q1 >> 8);
}

// ====================================================================================================================

__global__ void gaussian_filter_7x7_v6(int w, int h, const uchar *__restrict src, uchar *dst) // 32x8 blocks.
{
  // Position of the thread in the image.
  const int x = 1*(blockIdx.x*blockDim.x) + threadIdx.x;
  const int y = 2*(blockIdx.y*blockDim.y) + threadIdx.y;

  // Shared memory.
  __shared__ float smem_img[32][40];

  // Pixel to load.
  const int load_x = blockIdx.x*blockDim.x + 2*threadIdx.x - 4; // -4 for alignment (it should be -3).
  
  // Each thread loads 8 pixels.
  uchar2 p0 = in_img(load_x, y- 3, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y- 3)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p1 = in_img(load_x, y+ 5, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+ 5)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p2 = in_img(load_x, y+13, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+13)*w + load_x]) : make_uchar2(0, 0);
  uchar2 p3 = in_img(load_x, y+21, w, h) ? *reinterpret_cast<const uchar2*>(&src[(y+21)*w + load_x]) : make_uchar2(0, 0);

  // Store to shared memory.
  if( threadIdx.x < 20 )
  {
    reinterpret_cast<float2*>(smem_img[threadIdx.y+ 0])[threadIdx.x] = make_float2((float) p0.x, (float) p0.y);
    reinterpret_cast<float2*>(smem_img[threadIdx.y+ 8])[threadIdx.x] = make_float2((float) p1.x, (float) p1.y);
    reinterpret_cast<float2*>(smem_img[threadIdx.y+16])[threadIdx.x] = make_float2((float) p2.x, (float) p2.y);
    reinterpret_cast<float2*>(smem_img[threadIdx.y+24])[threadIdx.x] = make_float2((float) p3.x, (float) p3.y);
  }
  __syncthreads();
  
  // Compute the horizontal convolution.
  float n0[7], n1[7], n2[7], n3[7];
  for( int i = 0 ; i < 7 ; ++i )
  {
    n0[i] = smem_img[threadIdx.y+ 0][threadIdx.x + i+1]; // +1 because of alignment constraint when loading pixels.
    n1[i] = smem_img[threadIdx.y+ 8][threadIdx.x + i+1];
    n2[i] = smem_img[threadIdx.y+16][threadIdx.x + i+1];
    n3[i] = smem_img[threadIdx.y+24][threadIdx.x + i+1];
  }
  float p[4] = {0.0f};
  for( int i = 0 ; i < 7 ; ++i )
  {
    p[0] += gaussian_filter_fp32[i] * n0[i];
    p[1] += gaussian_filter_fp32[i] * n1[i];
    p[2] += gaussian_filter_fp32[i] * n2[i];
    p[3] += gaussian_filter_fp32[i] * n3[i];
  }
  __syncthreads();

  // Write the result back to shared memory.
  smem_img[threadIdx.y+ 0][threadIdx.x] = p[0];
  smem_img[threadIdx.y+ 8][threadIdx.x] = p[1];
  smem_img[threadIdx.y+16][threadIdx.x] = p[2];
  smem_img[threadIdx.y+24][threadIdx.x] = p[3];

  // Make sure the results are in SMEM.
  __syncthreads();

  // Compute the vertical convolution.
  float n[8];
  for( int i = 0 ; i < 8 ; ++i )
    n[i] = smem_img[2*threadIdx.y+i][threadIdx.x];
  float q0 = 0.0f, q1 = 0.0f;
  for( int i = 0 ; i < 7 ; ++i )
  {
    q0 += gaussian_filter_fp32[i] * n[i+0];
    q1 += gaussian_filter_fp32[i] * n[i+1];
  }

  // Where to write the result 2*(blockIdx.x*blockDim.x + threadIdx.y).
  const int write_y = y + threadIdx.y;

  // Write the pixels.
  if( in_img(x, write_y, w, h) )
    dst[write_y*w + x] = (uchar) ((int) q0 >> 8);
  if( in_img(x, write_y+1, w, h) )
    dst[(write_y+1)*w + x] = (uchar) ((int) q1 >> 8);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void sobel_filter_3x3_v0(int w, int h, const uchar *src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Early exit if the thread is not in the image.
  if( !in_img(x, y, w, h) )
    return;

  // Load the 8 neighbours and myself.
  int n[3][3];
  for( int j = -1 ; j <= 1 ; ++j )
    for( int i = -1 ; i <= 1 ; ++i )
      n[j+1][i+1] = in_img(x+i, y+j, w, h) ? (int) src[(y+j)*w + (x+i)] : 0;

  // Compute the convolution.
  int gx = 0, gy = 0;
  for( int j = 0 ; j < 3 ; ++j )
    for( int i = 0 ; i < 3 ; ++i )
    {
      gx += sobel_filter_x[j][i] * n[j][i];
      gy += sobel_filter_y[j][i] * n[j][i];
    }

  // The gradient.
  float grad = sqrtf((float) (gx*gx + gy*gy));

  // Store the result.
  dst[y*w + x] = (uchar) (grad >= 255.0f ? 255 : grad);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void sobel_filter_3x3_v1(int w, int h, const uchar *src, uchar *dst)
{
  // Position of the thread in the image.
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // Early exit if the thread is not in the image.
  if( !in_img(x, y, w, h) )
    return;

  // Load the 8 neighbours and myself.
  float n[3][3];
  for( int j = -1 ; j <= 1 ; ++j )
    for( int i = -1 ; i <= 1 ; ++i )
      n[j+1][i+1] = in_img(x+i, y+j, w, h) ? (float) src[(y+j)*w + (x+i)] : 0.0f;

  // Compute the convolution.
  float gx = 0.0f, gy = 0.0f;
  for( int j = 0 ; j < 3 ; ++j )
    for( int i = 0 ; i < 3 ; ++i )
    {
      gx = __fmaf_rd(sobel_filter_v1_x[j][i], n[j][i], gx);
      gy = __fmaf_rd(sobel_filter_v1_y[j][i], n[j][i], gy);
    }

  // The gradient.
  float grad = sqrtf((gx*gx + gy*gy));

  // Store the result.
  dst[y*w + x] = (uchar) (grad >= 255.0f ? 255 : grad);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

static void cuda_gaussian_filter(uchar *dst)
{
  //declare some timers for benchmarking the individual steps
  hipEvent_t totalStart;
  hipEvent_t totalEnd;
  hipEvent_t rtogStart;
  hipEvent_t rtogEnd;
  hipEvent_t gaussStart;
  hipEvent_t gaussEnd;
  hipEvent_t sobelStart;
  hipEvent_t sobelEnd;

  //initialize timers
  CHECK_CUDA(hipEventCreate(&totalStart));
  CHECK_CUDA(hipEventCreate(&totalEnd));
  CHECK_CUDA(hipEventCreate(&rtogStart));
  CHECK_CUDA(hipEventCreate(&rtogEnd));
  CHECK_CUDA(hipEventCreate(&gaussStart));
  CHECK_CUDA(hipEventCreate(&gaussEnd));
  CHECK_CUDA(hipEventCreate(&sobelStart));
  CHECK_CUDA(hipEventCreate(&sobelEnd));

// The size of the CUDA block/grid.
#  if OPTIMIZATION_STEP == 0x00
  #define OPTIMIZATION_DESC "Original version"
  dim3 block_dim(8, 8);
#elif OPTIMIZATION_STEP == 0x1a
  #define OPTIMIZATION_DESC "Block size 32x2 (It. 1, Eclipse Edition)"
  dim3 block_dim(32, 2);
#elif OPTIMIZATION_STEP == 0x1b
  #define OPTIMIZATION_DESC "Block size 8x16 (It 1, Visual Studio Edition)"
  dim3 block_dim(8, 16);
#elif OPTIMIZATION_STEP == 0x20
  #define OPTIMIZATION_DESC "Block size 32.4 (improve occupancy)"
  dim3 block_dim(32, 4);
#elif OPTIMIZATION_STEP == 0x30
  #define OPTIMIZATION_DESC "__launch_bounds__ to reduce registers (improve occupancy, It 2 Visual Studio Edition)"
  dim3 block_dim(32, 4);
#elif OPTIMIZATION_STEP == 0x40
  #define OPTIMIZATION_DESC "Using shared memory"
  dim3 block_dim(32, 4);
#elif OPTIMIZATION_STEP == 0x50
  #define OPTIMIZATION_DESC "Using read-only path (reduce pressure on Load-store unit)"
  dim3 block_dim(32, 4);
#else
  #if OPTIMIZATION_STEP == 0x60
    #define OPTIMIZATION_DESC "Using separable filter (reduce arithmetic intensity)"
  #elif OPTIMIZATION_STEP == 0x70
    #define OPTIMIZATION_DESC "Processing two elements per thread (increase ILP)"
  #elif OPTIMIZATION_STEP == 0x80
    #define OPTIMIZATION_DESC "Improved shared memory accesses (less bank conflicts)"
  #elif OPTIMIZATION_STEP == 0x90
    #define OPTIMIZATION_DESC "Using floats instead of ints (reducing arithmetic pipe pressure)"
  #elif OPTIMIZATION_STEP == 0x91
    #define OPTIMIZATION_DESC "Use floats and math intrinsics in sobel_filter (compile with --use_fast_math)"
  #else
    #define OPTIMIZATION_DESC "n/a"
  #endif
  dim3 block_dim(32, 8);
#endif
  dim3 grid_dim(round_up(g_data.img_w, block_dim.x), round_up(g_data.img_h, block_dim.y));

// The target.
  uchar *grayscale = g_data.show == SHOW_GRAYSCALE ? dst : g_data.img_grayscale;

#ifdef DO_WARMUP
  //run rgba_to_grayscale once to warm up everything to get better and more stable timings
  printf("Warming up...\n");
  rgba_to_grayscale_kernel_v0<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, g_data.img_rgba, grayscale);
  CHECK_CUDA(hipGetLastError());
#endif

  printf("Running version %2x: %s\n", OPTIMIZATION_STEP, OPTIMIZATION_DESC);

  //start total time timer
  CHECK_CUDA(hipEventRecord(totalStart));

  if( g_data.show == SHOW_RGBA )
  {
    CHECK_CUDA(hipMemcpy(dst, g_data.img_rgba, g_data.img_w*g_data.img_h*sizeof(uchar4), hipMemcpyDeviceToDevice));
    return;
  }

  // Convert from RGBA to Grayscale.
  dim3 block_dim_rgba(32, 8);
  
  CHECK_CUDA(hipEventRecord(rtogStart));
#if OPTIMIZATION_STEP == 0x91
  dim3 grid_dim_rgba(round_up(g_data.img_w, block_dim_rgba.x), round_up(g_data.img_h, block_dim_rgba.y)/2);
  rgba_to_grayscale_kernel_v1<<<grid_dim_rgba, block_dim_rgba>>>(g_data.img_w, g_data.img_h, g_data.img_rgba, grayscale);
#else
  dim3 grid_dim_rgba(round_up(g_data.img_w, block_dim_rgba.x), round_up(g_data.img_h, block_dim_rgba.y));
  rgba_to_grayscale_kernel_v0<<<grid_dim_rgba, block_dim_rgba>>>(g_data.img_w, g_data.img_h, g_data.img_rgba, grayscale);
#endif
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipEventRecord(rtogEnd));

  // Exit if we don't need more.
  if( g_data.show == SHOW_GRAYSCALE ) {
    CHECK_CUDA(hipEventRecord(totalEnd));
    printf("Times:\n");
    printf("-----------------------------------------------\n");
    printf("rgb_to_grayscale kernel           : %4.2f ms\n", getElapsedTimeInMS(rtogStart, rtogEnd));
    printf("gaussian_filter kernel            : n/a (didn't run)\n");
    printf("sobel_filter kernel               : n/a (didn't run)\n");
    printf("Total time in cuda_gaussian_filter: %4.2f ms\n", getElapsedTimeInMS(totalStart, totalEnd));
    printf("\n");
    return;
  }

  // The smoothed grayscale.
  uchar *smoothed_grayscale = grayscale;
  if( g_data.show & SHOW_SMOOTHED_GRAYSCALE )
  {
    smoothed_grayscale = (g_data.show & SHOW_EDGES) ? g_data.img_smoothed_grayscale : dst;
    
    CHECK_CUDA(hipEventRecord(gaussStart));

#if   OPTIMIZATION_STEP == 0x00
    gaussian_filter_7x7_v0<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
	gaussian_filter_7x7_v0<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x1a
    gaussian_filter_7x7_v0<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x1b
    gaussian_filter_7x7_v0<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x20
    gaussian_filter_7x7_v0<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x30
    gaussian_filter_7x7_v1<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x40
    gaussian_filter_7x7_v2<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x50
    gaussian_filter_7x7_v3<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x5a
    dim3 grid_dim0(grid_dim.x, grid_dim.y/2);
    gaussian_filter_7x7_v3_bis<<<grid_dim0, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x60
    gaussian_filter_7x7_v4<<<grid_dim, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x70
    dim3 grid_dim0(grid_dim.x, grid_dim.y/2);
    gaussian_filter_7x7_v5<<<grid_dim0, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#elif OPTIMIZATION_STEP == 0x80
    CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(gaussian_filter_7x7_v5), hipSharedMemBankSizeEightByte));
    dim3 grid_dim0(grid_dim.x, grid_dim.y/2);
    gaussian_filter_7x7_v5<<<grid_dim0, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#else
    CHECK_CUDA(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(gaussian_filter_7x7_v6), hipSharedMemBankSizeEightByte));
    dim3 grid_dim0(grid_dim.x, grid_dim.y/2);
    gaussian_filter_7x7_v6<<<grid_dim0, block_dim>>>(g_data.img_w, g_data.img_h, grayscale, smoothed_grayscale);
#endif
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipEventRecord(gaussEnd));
  }

  // Exit if we don't need more.
  if( !(g_data.show & SHOW_EDGES) ) {
    CHECK_CUDA(hipEventRecord(totalEnd));
    printf("Times:\n");
    printf("-----------------------------------------------\n");
    printf("rgb_to_grayscale kernel           : %4.2f ms\n", getElapsedTimeInMS(rtogStart, rtogEnd));
    if( g_data.show & SHOW_SMOOTHED_GRAYSCALE ) {
      printf("gaussian_filter kernel            : %4.2f ms\n", getElapsedTimeInMS(gaussStart, gaussEnd));
    } else {
      printf("gaussian_filter kernel            : n/a (didn't run)\n");
    }
    printf("sobel_filter kernel               : n/a (didn't run)\n");
    printf("Total time in cuda_gaussian_filter: %4.2f ms\n", getElapsedTimeInMS(totalStart, totalEnd));
    printf("\n");
    return;
  }

  dim3 block_dim_sobel(32, 8);
  CHECK_CUDA(hipEventRecord(sobelStart));
#if OPTIMIZATION_STEP == 0x91
  dim3 grid_dim_sobel(round_up(g_data.img_w, block_dim_sobel.x), round_up(g_data.img_h, block_dim_sobel.y));
  sobel_filter_3x3_v1<<<grid_dim_sobel, block_dim_sobel>>>(g_data.img_w, g_data.img_h, smoothed_grayscale, dst);
#else
  dim3 grid_dim_sobel(round_up(g_data.img_w, block_dim_sobel.x), round_up(g_data.img_h, block_dim_sobel.y));
  sobel_filter_3x3_v0<<<grid_dim_sobel, block_dim_sobel>>>(g_data.img_w, g_data.img_h, smoothed_grayscale, dst);
#endif
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipEventRecord(sobelEnd));

  CHECK_CUDA(hipEventRecord(totalEnd));
  printf("Times:\n");
  printf("-----------------------------------------------\n");
  printf("rgb_to_grayscale kernel           : %4.2f ms\n", getElapsedTimeInMS(rtogStart, rtogEnd));
  printf("gaussian_filter kernel            : %4.2f ms\n", getElapsedTimeInMS(gaussStart, gaussEnd));
  printf("sobel_filter kernel               : %4.2f ms\n", getElapsedTimeInMS(sobelStart, sobelEnd));
  printf("Total time in cuda_gaussian_filter: %4.2f ms\n", getElapsedTimeInMS(totalStart, totalEnd));
  printf("\n");
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef WITH_OPENGL
static void display()
{
  // Map the pixel buffer object.
  uchar *dst;
  CHECK_CUDA(hipGraphicsMapResources(1, &g_data.img_cuda_pbo, 0));
  CHECK_CUDA(hipGraphicsResourceGetMappedPointer((void **) &dst, NULL, g_data.img_cuda_pbo));
  cuda_gaussian_filter(dst);
  CHECK_CUDA(hipGraphicsUnmapResources(1, &g_data.img_cuda_pbo, 0) );

  // Prepare the texture.
  CHECK_OPENGL(glClear(GL_COLOR_BUFFER_BIT));
  CHECK_OPENGL(glBindTexture(GL_TEXTURE_2D, g_data.img_tex));
  CHECK_OPENGL(glBindBuffer(GL_PIXEL_UNPACK_BUFFER, g_data.img_pbo));
  if( g_data.show == SHOW_RGBA )
    CHECK_OPENGL(glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, g_data.img_w, g_data.img_h, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL));
  else
    CHECK_OPENGL(glTexImage2D(GL_TEXTURE_2D, 0, GL_LUMINANCE, g_data.img_w, g_data.img_h, 0, GL_LUMINANCE, GL_UNSIGNED_BYTE, NULL));
  CHECK_OPENGL(glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0));

  // Set the texture parameters.
  CHECK_OPENGL(glDisable(GL_DEPTH_TEST));
  CHECK_OPENGL(glEnable(GL_TEXTURE_2D));
  CHECK_OPENGL(glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR));
  CHECK_OPENGL(glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR));
  CHECK_OPENGL(glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT));
  CHECK_OPENGL(glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT));

  // Show the image.
  glBegin(GL_QUADS);
  glVertex2f  (0.0f, 0.0f);
  glTexCoord2f(0.0f, 0.0f);
  glVertex2f  (0.0f, 1.0f);
  glTexCoord2f(1.0f, 0.0f);
  glVertex2f  (1.0f, 1.0f);
  glTexCoord2f(1.0f, 1.0f);
  glVertex2f  (1.0f, 0.0f);
  glTexCoord2f(0.0f, 1.0f);
  CHECK_OPENGL(glEnd());
  CHECK_OPENGL(glBindTexture(GL_TEXTURE_2D, 0));
  glutSwapBuffers();
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
  if( key == 27 || key == 'Q' || key == 'q' )
    exit(1);
  if( key == 'R' || key == 'r' )
    g_data.show = SHOW_RGBA;
  if( key == 'G' || key == 'g' )
   g_data.show = (g_data.show | SHOW_GRAYSCALE) ^ SHOW_SMOOTHED_GRAYSCALE;
  if( key == 'E' || key == 'e' )
    g_data.show = (g_data.show | SHOW_GRAYSCALE) ^ SHOW_EDGES;
  glutPostRedisplay();
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void reshape(int x, int y)
{
  CHECK_OPENGL(glViewport(0, 0, x, y));
  CHECK_OPENGL(glMatrixMode(GL_PROJECTION));
  CHECK_OPENGL(glLoadIdentity());
  CHECK_OPENGL(glOrtho(0, 1, 0, 1, 0, 1));
  CHECK_OPENGL(glMatrixMode(GL_MODELVIEW));
  CHECK_OPENGL(glLoadIdentity());
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void init_gl(int *argc, char **argv)
{
  glutInit(argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  float ratio = (float) g_data.img_h / g_data.img_w;
  glutInitWindowSize(min(g_data.img_w, 1024), min(g_data.img_h, (int) (1024*ratio)));
  glutCreateWindow("CUDA Gaussian Filter");

  glewInit();

  if( !glewIsSupported("GL_VERSION_1_5 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object GL_ARB_texture_float") )
  {
    fprintf(stderr, "Error: failed to get minimal extensions for demo\n");
    fprintf(stderr, "This sample requires:\n");
    fprintf(stderr, "  OpenGL version 1.5\n");
    fprintf(stderr, "  GL_ARB_vertex_buffer_object\n");
    fprintf(stderr, "  GL_ARB_pixel_buffer_object\n");
    exit(1);
  }

  // Create the PBO.
  CHECK_OPENGL(glGenBuffers(1, &g_data.img_pbo));
  CHECK_OPENGL(glBindBuffer(GL_PIXEL_UNPACK_BUFFER, g_data.img_pbo));
  CHECK_OPENGL(glBufferData(GL_PIXEL_UNPACK_BUFFER, g_data.img_w*g_data.img_h*sizeof(uchar4), NULL, GL_STREAM_DRAW));
  CHECK_OPENGL(glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0));

  // Register this buffer object with CUDA.
  CHECK_CUDA(hipGraphicsGLRegisterBuffer(&g_data.img_cuda_pbo, g_data.img_pbo, cudaGraphicsMapFlagsWriteDiscard));

  // Create the OpenGL texture.
  CHECK_OPENGL(glGenTextures(1, &g_data.img_tex));
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void cleanup(void)
{
  hipGraphicsUnregisterResource(g_data.img_cuda_pbo);

  glDeleteBuffers(1, &g_data.img_pbo);
  glDeleteTextures(1, &g_data.img_tex);
  CHECK_CUDA(hipFree(g_data.img_rgba));
  CHECK_CUDA(hipFree(g_data.img_grayscale));
  CHECK_CUDA(hipFree(g_data.img_smoothed_grayscale));
}
#endif

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void load_image(const char *filename)
{
  const int HEADER_SIZE = 64;

  // Open the source file.
  FILE *fp = NULL;
  CHECK_WITH_MSG(fp = fopen(filename, "rb"), "Cannot open file");

  // Read header.
  char header[HEADER_SIZE];
  CHECK(fgets(header, HEADER_SIZE, fp));
  
  // Number of channels. Must be 3.
  CHECK(!strncmp(header, "P6", 2));

  // Skip the comments.
  while( fgets(header, HEADER_SIZE, fp) && header[0] == '#' )
    ;

  // Parse the header.
  int width = 0, height = 0;
  CHECK(sscanf(header, "%u %u", &width, &height) == 2);
  printf("Image %s: w=%4d x h=%4d\n", filename, width, height);

  // Ignore the max value.
  CHECK(fgets(header, HEADER_SIZE, fp));

  // Read the pixels.
  int size_in_bytes = 3*width*height*sizeof(uchar);
  uchar *img_rgb = (uchar*) malloc(size_in_bytes);  
  CHECK(img_rgb);
  CHECK(fread(img_rgb, sizeof(uchar), 3*width*height, fp) == 3*width*height);

  // Close the file.
  fclose(fp);

  // Create the RGBA image on the host.
  size_in_bytes = width*height*sizeof(uchar4);
#ifdef WITH_UVM
  uchar4 *img_rgba;
  CHECK_CUDA(hipMallocManaged((void**)&img_rgba, size_in_bytes));
#else
  uchar4 *img_rgba = (uchar4*)malloc(size_in_bytes);
#endif
  CHECK(img_rgba);
  for( int i = 0 ; i < width*height ; ++i )
    img_rgba[i] = make_uchar4(img_rgb[3*i+0], img_rgb[3*i+1], img_rgb[3*i+2], 0);
  free(img_rgb);

  // Setup the global data.
  g_data.img_w = width;
  g_data.img_h = height;

#ifdef WITH_UVM
  g_data.img_rgba = img_rgba;
  size_in_bytes = width*height*sizeof(uchar);
  CHECK_CUDA(hipMallocManaged((void**)&g_data.img_grayscale, size_in_bytes));
  CHECK_CUDA(hipMallocManaged((void**)&g_data.img_smoothed_grayscale, size_in_bytes));
#else
  // Allocate CUDA memory.
  CHECK_CUDA(hipMalloc((void**) &g_data.img_rgba, size_in_bytes));
  CHECK_CUDA(hipMemcpy(g_data.img_rgba, img_rgba, size_in_bytes, hipMemcpyHostToDevice));
  free(img_rgba);

  // Allocate other temp buffers.
  size_in_bytes = width*height*sizeof(uchar);
  CHECK_CUDA(hipMalloc((void**) &g_data.img_grayscale, size_in_bytes));
  CHECK_CUDA(hipMalloc((void**) &g_data.img_smoothed_grayscale, size_in_bytes));
#endif
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
  // Parse the input argument if any.
#ifdef WITH_OPENGL
  bool use_opengl = true;
  if( argc == 3 && !strcmp(argv[1], "-no-opengl") )
    use_opengl = false;
  else
#endif // WITH_OPENGL
  if( argc < 2 || argc > 4 )
  {
#ifdef WITH_OPENGL
    printf("Usage: %s [-no-opengl] [device-num] FILENAME.ppm\n", argv[0]);
#else
    printf("Usage: %s [device-num] FILENAME.ppm\n", argv[0]);
#endif
    exit(1);
  }

  //we abuse the fact that atoi returns 0 on a non-integer string
  CHECK_CUDA(hipSetDevice(atoi(argv[argc-2])));
  int deviceNum;
  CHECK_CUDA(hipGetDevice(&deviceNum));
  struct hipDeviceProp_t deviceProps;
  CHECK_CUDA(hipGetDeviceProperties(&deviceProps, deviceNum));
  printf("Using device #%d: %s (cc %d.%d)\n", deviceNum, deviceProps.name, deviceProps.major, deviceProps.minor);

  // Clear the global data.
  memset(&g_data, 0, sizeof(g_data));

  // Read the input image.
  load_image(argv[argc-1]);

  // Initialize OpenGL if needed.
#ifdef WITH_OPENGL
  if( use_opengl )
  {
    g_data.show = SHOW_RGBA;

    init_gl(&argc, argv);
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);

    // If code is not printing the USage, then we execute this path.
    printf("R: RGBA image\n");
    printf("G: Toggle Gaussian filter\n");
    printf("E: Toggle Sobel filter\n");
    printf("Q: Quit\n");
    fflush(stdout);
    atexit(cleanup);
    glutMainLoop();
  }
  else
#endif // WITH_OPENGL
  {
    g_data.show = SHOW_GRAYSCALE | SHOW_SMOOTHED_GRAYSCALE | SHOW_EDGES;

    uchar *dst = NULL;
#ifdef WITH_UVM
    CHECK_CUDA(hipMallocManaged((void**) &dst, g_data.img_w*g_data.img_h*sizeof(uchar)));
#else
	CHECK_CUDA(hipMalloc((void**)&dst, g_data.img_w*g_data.img_h*sizeof(uchar)));
#endif
	cuda_gaussian_filter(dst);
    CHECK_CUDA(hipFree(dst));
  }

  // Free CUDA resources.
  CHECK_CUDA(hipFree(g_data.img_rgba));
  CHECK_CUDA(hipFree(g_data.img_grayscale));
  CHECK_CUDA(hipFree(g_data.img_smoothed_grayscale));

  CHECK_CUDA(hipDeviceReset());
  return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

